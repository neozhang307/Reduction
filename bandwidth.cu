#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>
#include<math.h>
namespace cg = cooperative_groups;



__global__ void reduce_basic(double *g_idata, double *g_odata, unsigned int n)
{
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;	
   	unsigned int i=blockIdx.x*blockDim.x*2 + threadIdx.x;	
    unsigned int gridSize = blockDim.x*gridDim.x*2;
    // printf("gridsize:%d\tblockdim:%d\tgridDim:%dn",gridSize, blockDim.x, gridDim.x);
    double sum=0;
    while (i < n)
    {
    	sum+=g_idata[i];
    	// if(i+blockDim.x<n)
    	sum+=g_idata[i+blockDim.x];
    	i += gridSize;
    }
    g_odata[tid]=sum;
}


__global__ void copy_basic(double *g_idata, double *g_odata, unsigned int n)
{
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x; 
    unsigned int i=blockIdx.x*blockDim.x*2 + threadIdx.x; 
    unsigned int gridSize = blockDim.x*gridDim.x*2;
    // printf("gridsize:%d\tblockdim:%d\tgridDim:%dn",gridSize, blockDim.x, gridDim.x);
    double tmp1;
    double tmp2;
    while (i < n)
    {
      tmp1=g_idata[i];
      tmp2=g_idata[i+blockDim.x];
      g_odata[i]=tmp1;
      g_odata[tid]=tmp2;
      i += gridSize;
    }
}

 #define single_test(func) \
 do{\
	double* h_input;\
	double* d_input;\
	double* h_output;\
	double* d_output;\
	hipEvent_t start, end;\
	hipEventCreate(&start);\
	hipEventCreate(&end);\
\
	hipHostAlloc((void**)& h_input, size*sizeof(double), hipHostMallocDefault);\
	hipHostAlloc((void**)& h_output, size*sizeof(double), hipHostMallocDefault);\
	hipMalloc((void**)&d_input, size*sizeof(double));\
	hipMalloc((void**)&d_output, size*sizeof(double));\
\
	for(int i=0; i<size; i++)\
	{\
		h_input[i]=1;\
	}\
	hipMemcpy(d_input, h_input, size*sizeof(double), hipMemcpyHostToDevice);\
	hipEventRecord(start);\
	func<<<block*smx_count,thread>>>(d_input,d_output,size);\
	hipEventRecord(end);\
	hipDeviceSynchronize();\
	hipError_t e=hipGetLastError();\
	if(e!=hipSuccess) \
    { \
        printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));\
    }\
	hipEventElapsedTime(&millisecond,start,end);\
	hipHostFree(h_input);\
	hipHostFree(h_output);\
	hipFree(d_input);\
	hipFree(d_output);\
	hipEventDestroy(start);\
	hipEventDestroy(end);\
	hipDeviceReset();\
}while(0)\

 #define single_block_test(func) \
 do{\
  double* h_input;\
  double* d_input;\
  double* h_output;\
  double* d_output;\
  hipEvent_t start, end;\
  hipEventCreate(&start);\
  hipEventCreate(&end);\
\
  hipHostAlloc((void**)& h_input, size*sizeof(double), hipHostMallocDefault);\
  hipHostAlloc((void**)& h_output, size*sizeof(double), hipHostMallocDefault);\
  hipMalloc((void**)&d_input, size*sizeof(double));\
  hipMalloc((void**)&d_output, size*sizeof(double));\
\
  for(int i=0; i<size; i++)\
  {\
    h_input[i]=1;\
  }\
  hipMemcpy(d_input, h_input, size*sizeof(double), hipMemcpyHostToDevice);\
  hipEventRecord(start);\
  func<<<1,thread>>>(d_input,d_output,size);\
  hipEventRecord(end);\
  hipDeviceSynchronize();\
  hipError_t e=hipGetLastError();\
  if(e!=hipSuccess) \
    { \
        printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));\
    }\
  hipEventElapsedTime(&millisecond,start,end);\
  hipHostFree(h_input);\
  hipHostFree(h_output);\
  hipFree(d_input);\
  hipFree(d_output);\
  hipEventDestroy(start);\
  hipEventDestroy(end);\
  hipDeviceReset();\
}while(0)\

 #define single_warp_test() \
    do{\
    double* h_input;\
    double* d_input;\
    double* h_output;\
    double* d_output;\
    unsigned int* d_time_stamp;\
    unsigned int* h_time_stamp=(unsigned int*)malloc(sizeof(unsigned int)*thread*2/32);\
      \
    hipHostAlloc((void**)& h_input, smsize*sizeof(double), hipHostMallocDefault);\
    hipHostAlloc((void**)& h_output, smsize*sizeof(double), hipHostMallocDefault);\
    hipMalloc((void**)&d_input, smsize*sizeof(double));\
    hipMalloc((void**)&d_output, smsize*sizeof(double));\
    hipMalloc((void**)&d_time_stamp, thread*2/32*sizeof(unsigned int));\
  \
    for(int i=0; i<smsize; i++)\
    {\
      h_input[i]=1;\
    }\
    hipMemcpy(d_input, h_input, smsize*sizeof(double), hipMemcpyHostToDevice);\
    reduce_basic_warp<<<1,thread>>>(d_input,d_output,smsize,executor_num,d_time_stamp);\
    hipMemcpy(h_time_stamp, d_time_stamp, thread*2/32*sizeof(unsigned int), hipMemcpyDeviceToHost);\
    hipDeviceSynchronize();\
    hipError_t e=hipGetLastError();\
    if(e!=hipSuccess) \
      { \
          printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));\
      }\
      unsigned int start=h_time_stamp[0];\
      unsigned int end=h_time_stamp[1];\
      for(int i=1; i<thread/32; i++)\
      {\
        start=min(start,h_time_stamp[i*2]);\
        end=max(end,h_time_stamp[i*2+1]);\
      }\
      latency_cycle=end-start;\
      free(h_time_stamp);\
    hipHostFree(h_input);\
    hipFree(d_time_stamp);\
    hipHostFree(h_output);\
    hipFree(d_input);\
    hipFree(d_output);\
    hipDeviceReset();\
  }while(0);\

#define TEST_TIME 21
#define SKIP 1

int main()
{
	unsigned int size=500000000;
 	hipDeviceProp_t deviceProp;
  	hipSetDevice(0);
  	hipGetDeviceProperties(&deviceProp, 0);
  	unsigned int smx_count = deviceProp.multiProcessorCount;

    unsigned int base=2000;
    size=base*smx_count*2048*2;
    
    float millisecond;
    float lats[TEST_TIME];

  	
   //  unsigned int block=1;
  	// unsigned int thread = 1024;

    for(int block=1; block<=64; block*=2)
    { 
      for(int thread=32; thread<=1024; thread*=2)
      {
        for(int i=0; i<TEST_TIME; i++)
        {
           single_test(reduce_basic);
           lats[i]=millisecond;
        }
        millisecond=0;
       for(int i=SKIP; i<TEST_TIME; i++)
        {
          millisecond+=lats[i];
        }
        millisecond=millisecond/(TEST_TIME-SKIP);
        printf("block/SM %d thread %d totalsize %d time: %f ms speed: %f GB/s\n",
          block,thread, size,
          millisecond, size*sizeof(double)/millisecond/1000/1000);
      }
    }

    size=size/smx_count*2;
      for(int thread=32; thread<=1024; thread*=2)
      {
        for(int i=0; i<TEST_TIME; i++)
        {
           single_block_test(reduce_basic);
           lats[i]=millisecond;
        }
        millisecond=0;
       for(int i=SKIP; i<TEST_TIME; i++)
        {
          millisecond+=lats[i];
        }
        millisecond=millisecond/(TEST_TIME-SKIP);
        printf("block/GPU %d thread %d totalsize %d time: %f ms speed: %f GB/s\n",
          1,thread, size,
          millisecond, size*sizeof(double)/millisecond/1000/1000);
      }



// printf("thread %d, executer %d, smsize %d, time: %d cycle speed: %f GB/s\n",
//       thread, executor_num, smsize, latency, (double)smsize*sizeof(double)/latency);
    }