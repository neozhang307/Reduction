#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>
#include<math.h>
namespace cg = cooperative_groups;



__global__ void reduce_basic(double *g_idata, double *g_odata, unsigned int n)
{
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;	
   	unsigned int i=blockIdx.x*blockDim.x*2 + threadIdx.x;	
    unsigned int gridSize = blockDim.x*gridDim.x*2;
    // printf("gridsize:%d\tblockdim:%d\tgridDim:%dn",gridSize, blockDim.x, gridDim.x);
    double sum=0;
    while (i < n)
    {
    	sum+=g_idata[i];
    	// if(i+blockDim.x<n)
    	sum+=g_idata[i+blockDim.x];
    	i += gridSize;
    }
    g_odata[tid]=sum;
}

__global__ void reduce_basic_warp(double *g_idata, double *g_odata,
  unsigned int n, unsigned int executor_num, unsigned int *time_stamp)
{
    unsigned int tid = threadIdx.x; 
    unsigned int i=threadIdx.x; 
    unsigned int warp_id=i/32;
    cg::thread_block cta = cg::this_thread_block();
    unsigned int  start,stop;
    __shared__ double sdata[4*1024];
    double sum=0;
    for(int i=tid; i<n; i+=blockDim.x)
    {
      sum=g_idata[i];
      sdata[i]=sum;
    }
    cg::sync(cta);
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");
    if(tid<executor_num)
    {
      while (i < n)
      {
        sum+=sdata[i];
        i += executor_num;
      }
    }
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
    g_odata[tid]=sum;
    if(i%32==0)
    {
        time_stamp[warp_id*2]=start;
        time_stamp[warp_id*2+1]=stop;
    }
}

__global__ void copy_basic(double *g_idata, double *g_odata, unsigned int n)
{
    unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x; 
    unsigned int i=blockIdx.x*blockDim.x*2 + threadIdx.x; 
    unsigned int gridSize = blockDim.x*gridDim.x*2;
    // printf("gridsize:%d\tblockdim:%d\tgridDim:%dn",gridSize, blockDim.x, gridDim.x);
    double tmp1;
    double tmp2;
    while (i < n)
    {
      tmp1=g_idata[i];
      tmp2=g_idata[i+blockDim.x];
      g_odata[i]=tmp1;
      g_odata[tid]=tmp2;
      i += gridSize;
    }
}

 #define single_test(func) \
 do{\
	double* h_input;\
	double* d_input;\
	double* h_output;\
	double* d_output;\
	hipEvent_t start, end;\
	hipEventCreate(&start);\
	hipEventCreate(&end);\
\
	hipHostAlloc((void**)& h_input, size*sizeof(double), hipHostMallocDefault);\
	hipHostAlloc((void**)& h_output, size*sizeof(double), hipHostMallocDefault);\
	hipMalloc((void**)&d_input, size*sizeof(double));\
	hipMalloc((void**)&d_output, size*sizeof(double));\
\
	for(int i=0; i<size; i++)\
	{\
		h_input[i]=1;\
	}\
	hipMemcpy(d_input, h_input, size*sizeof(double), hipMemcpyHostToDevice);\
	hipEventRecord(start);\
	func<<<block*smx_count,thread>>>(d_input,d_output,size);\
	hipEventRecord(end);\
	hipDeviceSynchronize();\
	hipError_t e=hipGetLastError();\
	if(e!=hipSuccess) \
    { \
        printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));\
    }\
	hipEventElapsedTime(&millisecond,start,end);\
	hipHostFree(h_input);\
	hipHostFree(h_output);\
	hipFree(d_input);\
	hipFree(d_output);\
	hipEventDestroy(start);\
	hipEventDestroy(end);\
	hipDeviceReset();\
}while(0)\

 #define single_block_test(func) \
 do{\
  double* h_input;\
  double* d_input;\
  double* h_output;\
  double* d_output;\
  hipEvent_t start, end;\
  hipEventCreate(&start);\
  hipEventCreate(&end);\
\
  hipHostAlloc((void**)& h_input, size*sizeof(double), hipHostMallocDefault);\
  hipHostAlloc((void**)& h_output, size*sizeof(double), hipHostMallocDefault);\
  hipMalloc((void**)&d_input, size*sizeof(double));\
  hipMalloc((void**)&d_output, size*sizeof(double));\
\
  for(int i=0; i<size; i++)\
  {\
    h_input[i]=1;\
  }\
  hipMemcpy(d_input, h_input, size*sizeof(double), hipMemcpyHostToDevice);\
  hipEventRecord(start);\
  func<<<1,thread>>>(d_input,d_output,size);\
  hipEventRecord(end);\
  hipDeviceSynchronize();\
  hipError_t e=hipGetLastError();\
  if(e!=hipSuccess) \
    { \
        printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));\
    }\
  hipEventElapsedTime(&millisecond,start,end);\
  hipHostFree(h_input);\
  hipHostFree(h_output);\
  hipFree(d_input);\
  hipFree(d_output);\
  hipEventDestroy(start);\
  hipEventDestroy(end);\
  hipDeviceReset();\
}while(0)\

 #define single_warp_test() \
    do{\
    double* h_input;\
    double* d_input;\
    double* h_output;\
    double* d_output;\
    unsigned int* d_time_stamp;\
    unsigned int* h_time_stamp=(unsigned int*)malloc(sizeof(unsigned int)*thread*2/32);\
      \
    hipHostAlloc((void**)& h_input, smsize*sizeof(double), hipHostMallocDefault);\
    hipHostAlloc((void**)& h_output, smsize*sizeof(double), hipHostMallocDefault);\
    hipMalloc((void**)&d_input, smsize*sizeof(double));\
    hipMalloc((void**)&d_output, smsize*sizeof(double));\
    hipMalloc((void**)&d_time_stamp, thread*2/32*sizeof(unsigned int));\
  \
    for(int i=0; i<smsize; i++)\
    {\
      h_input[i]=1;\
    }\
    hipMemcpy(d_input, h_input, smsize*sizeof(double), hipMemcpyHostToDevice);\
    reduce_basic_warp<<<1,thread>>>(d_input,d_output,smsize,executor_num,d_time_stamp);\
    hipMemcpy(h_time_stamp, d_time_stamp, thread*2/32*sizeof(unsigned int), hipMemcpyDeviceToHost);\
    hipDeviceSynchronize();\
    hipError_t e=hipGetLastError();\
    if(e!=hipSuccess) \
      { \
          printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));\
      }\
      unsigned int start=h_time_stamp[0];\
      unsigned int end=h_time_stamp[1];\
      for(int i=1; i<thread/32; i++)\
      {\
        start=min(start,h_time_stamp[i*2]);\
        end=max(end,h_time_stamp[i*2+1]);\
      }\
      latency_cycle=end-start;\
      free(h_time_stamp);\
    hipHostFree(h_input);\
    hipFree(d_time_stamp);\
    hipHostFree(h_output);\
    hipFree(d_input);\
    hipFree(d_output);\
    hipDeviceReset();\
  }while(0);\

#define TEST_TIME 21
#define SKIP 1

int main()
{
	unsigned int size=500000000;
 	hipDeviceProp_t deviceProp;
  	hipSetDevice(0);
  	hipGetDeviceProperties(&deviceProp, 0);
  	unsigned int smx_count = deviceProp.multiProcessorCount;

    unsigned int base=2000;
    size=base*smx_count*2048*2;
    
    float millisecond;
    float lats[TEST_TIME];

  	
   //  unsigned int block=1;
  	// unsigned int thread = 1024;

    for(int block=1; block<=64; block*=2)
    { 
      for(int thread=32; thread<=1024; thread*=2)
      {
        for(int i=0; i<TEST_TIME; i++)
        {
           single_test(reduce_basic);
           lats[i]=millisecond;
        }
        millisecond=0;
       for(int i=SKIP; i<TEST_TIME; i++)
        {
          millisecond+=lats[i];
        }
        millisecond=millisecond/(TEST_TIME-SKIP);
        printf("block/SM %d thread %d totalsize %d time: %f ms speed: %f GB/s\n",
          block,thread, size,
          millisecond, size*sizeof(double)/millisecond/1000/1000);
      }
    }

    size=size/smx_count*2;
      for(int thread=32; thread<=1024; thread*=2)
      {
        for(int i=0; i<TEST_TIME; i++)
        {
           single_block_test(reduce_basic);
           lats[i]=millisecond;
        }
        millisecond=0;
       for(int i=SKIP; i<TEST_TIME; i++)
        {
          millisecond+=lats[i];
        }
        millisecond=millisecond/(TEST_TIME-SKIP);
        printf("block/GPU %d thread %d totalsize %d time: %f ms speed: %f GB/s\n",
          1,thread, size,
          millisecond, size*sizeof(double)/millisecond/1000/1000);
      }


    unsigned int smsize=4*1024;
    unsigned int thread=1024;
    // unsigned int executor_num=32;
    unsigned int latency_cycle;
    unsigned int lat_cycle_s[TEST_TIME];
    float latency_tmp;
    for(unsigned int executor_num =1; executor_num<=32; executor_num++)
    {
        // for(thread=32; thread<=1024; thread*=2)
        // {
          for(int i=0; i<TEST_TIME; i++)
          {
             single_warp_test();
             lat_cycle_s[i]=latency_cycle;
          }      
          latency_tmp=0;
         for(int i=SKIP; i<TEST_TIME; i++)
          {
            latency_tmp+=lat_cycle_s[i];
          }
          latency_tmp=latency_tmp/(TEST_TIME-SKIP);
          printf("thread %d, executer %d, smsize %d, time: %f cycle speed: %f Byte/cycle\n",
                thread, executor_num, smsize, latency_tmp, (double)smsize*sizeof(double)/latency_tmp);
        // }
    }
    for(unsigned int executor_num =32; executor_num<=1024; executor_num*=2)
    {
        // for(thread=32; thread<=1024; thread*=2)
        // {
          if(executor_num>thread)continue;
          for(int i=0; i<TEST_TIME; i++)
          {
             single_warp_test();
             lat_cycle_s[i]=latency_cycle;
          }      
          // single_warp_test();
          latency_tmp=0;
         for(int i=SKIP; i<TEST_TIME; i++)
          {
            latency_tmp+=lat_cycle_s[i];
          }
          latency_tmp=latency_tmp/(TEST_TIME-SKIP);
          printf("thread %d, executer %d, smsize %d, time: %f cycle speed: %f Byte/cycle\n",
                thread, executor_num, smsize, latency_tmp, (double)smsize*sizeof(double)/latency_tmp);
        // }
    }

// printf("thread %d, executer %d, smsize %d, time: %d cycle speed: %f GB/s\n",
//       thread, executor_num, smsize, latency, (double)smsize*sizeof(double)/latency);
    }